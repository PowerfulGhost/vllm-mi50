#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

#include "ggml-common.h"
#include "vecdotq.cuh"
#include "dequantize.cuh"
#include "mmvq.cuh"
#include "mmq.cuh"
#include "moe.cuh"

#include "mmq_mi50.cuh"
